#include "hip/hip_runtime.h"
#include	<stdio.h>
#include 	<stdlib.h>
#include 	<string.h>
#include	<math.h>
#include	<iostream>
#include	<fstream>
#include    "sim_run.h"
#include    "calc_neighbor_list.h"
#include    "random_mars.h"
//#include    "hip/hip_runtime.h"
#include    "hiprand/hiprand_kernel.h"
#include <map>
#include <vector>
using namespace std;

double verify_f(double** f) {
    double sum_f = 0;
    for (int j = 0; j < N; j++) {
        sum_f += f[j][0];
        sum_f += f[j][1];
        sum_f += f[j][2];
    }
    return sum_f;
}

#if __CUDA_ARCH__ < 600
__device__ double atomicAdd_30(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}
#endif

void cpu_dpd(double** r, double**f, double** v, const char* select, RanMars* random, OUTPUT_struct* output_info, double len_cell, double len_x, double len_y, double len_z) {
        ofstream outputfile;
        outputfile.open("dump_cpu.md", ios::out);
        //init force compute
        if (!strcmp(select, "sijun"))
            compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
        else if (!strcmp(select, "base")) 
            compute_force_std(r, v, f, random, N);
        //writeDump(outputfile, r, v, 0);
        int ntimestep = 100;//5000;
        double m = 1.0;
        for (int i = 0; i <= ntimestep; i++) {

            //half integration
            for(int j = 0; j < N; j++) {
                v[j][0] += 0.5 * f[j][0] * dt;
                v[j][1] += 0.5 * f[j][1] * dt;
                v[j][2] += 0.5 * f[j][2] * dt;

                r[j][0] += v[j][0] * dt;
                r[j][1] += v[j][1] * dt;
                r[j][2] += v[j][2] * dt;
            }
            pbc(r); 
            //force computation
            clear_force(f, N);
            if (!strcmp(select, "sijun")) {
                compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
                int partc_num_def = 4000;
                output_info =  func_partc_incell_stat(r, partc_num_def, len_cell, len_x, len_y, len_z);
            }
            else if (!strcmp(select, "base"))
                compute_force_std(r, v, f, random, N);
            //full integration
            for(int j = 0; j < N; j++) {
                v[j][0] += 0.5 * f[j][0] * dt;
                v[j][1] += 0.5 * f[j][1] * dt;
                v[j][2] += 0.5 * f[j][2] * dt;
            }
             if(i % 1 == 0) {
                double ke = computeKE(v);
                cout << i << " temp is " << ke * 2 / (3 * 4000 * 1) << endl;
                writeDump(outputfile, r, f, i);
                //cout << "verify force on cpu at step " << i << " is " << verify_f(f) << endl;
            }
        }
}

__global__ void print_kernel() {
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

__global__ void init(double *r, double *v, double *f) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < N) {
            int i;
             for(i = 0; i < 3; i++) {
                v[id * 3 + i] += 0.5 * f[id * 3 + i] * dt;
                r[id * 3 + i] += v[id * 3 + i] * dt;
            } 
            // do periodic boundary condition
            for (i = 0; i < 3; i++) {
                if (r[id * 3 + i] < 0)
                    r[id * 3 + i] += 10;
                else if (r[id * 3 + i] > 10)
                    r[id * 3 + i] -= 10;
            }
    }
}

__global__ void f_clear(double *f) {
     int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < N) {
           //force computation
            int i;
            for (i = 0; i < 3; i++)
                f[id * 3 + i] = 0;
 }  
}

__global__ void setup_kernel(hiprandState *state) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < N)
        hiprand_init(34387, id, 0, &state[id]);
}

__global__ void iteration(hiprandState *state, double* r, double* f, double* v, int* cell_list, int* cell_list_count, int avg_num_cell, int * dev_ts) {
    //printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
    int id = threadIdx.x + blockDim.x * blockIdx.x;
        //printf("blockIdx.x %d threadIdx.x %d\n", blockIdx.x, threadIdx.x); 
    if (id < N) {
            hiprandState localState = state[id];
            //hiprand_init(34387+id, id, 0, &state[id]);

            int i;
            if (id == 0) {
                double sum_f = 0;
                for (int j = 0; j < 3 * N; j++) {
                    sum_f += f[j];
                }
                //printf("sum f is %f \n", sum_f);
            }
            double len_cell = 2.0;
            double len_x = 10.0;
            double len_y = 10.0;
            double len_z = 10.0;
            double m = 1.0;
            int num_cx, num_cy, num_cz;
            num_cx = (int) floor(len_x/len_cell);
            num_cy = (int) floor(len_y/len_cell);
            num_cz = (int) floor(len_z/len_cell);
            //printf("number of cell is %d \n", num_cx * num_cy * num_cz);
            double x = r[id * 3];
            double y = r[id * 3 + 1];
            double z = r[id * 3 + 2];

            // which cell in 3d the particle is in
            int idx = (int) floor(x / len_cell);
            int idy = (int) floor(y / len_cell);
            int idz = (int) floor(z / len_cell);
                        // loop through 27 boxes
            for (int n = -1; n < 2; n++) {
                for (int m = -1; m < 2; m++) {
                    for (int l = -1; l < 2; l++) {
                        int newidz = (idz + n + num_cz) % num_cz;
                        int newidy = (idy + m + num_cy) % num_cy;
                        int newidx = (idx + l + num_cx) % num_cx;
                        int cell_id = newidz + newidy * num_cz + newidx * num_cy * num_cz;
                        if (id == 3999) {
                            //printf("l %d m %d n %d cell_id %d \n", l, m, n, cell_id);
                        }
                        int num_neigh_particle = cell_list_count[cell_id];
                         if (id == 3999) {
                        //    printf("l %d m %d n %d num_neigh_particle %d \n", l, m, n, num_neigh_particle);
                        }
                        for (int j = 0; j < num_neigh_particle; j++) {
                            int part_id = cell_list[cell_id * avg_num_cell + j];
                            if (part_id > id) {
                                if (part_id == id) continue; 
                                double delx, dely, delz;
                                delx = x - r[part_id * 3];
                                dely = y - r[part_id * 3 + 1];
                                delz = z - r[part_id * 3 + 2];

                               if (delx < -5)
                                    delx = delx + 10;
                                else if (delx > 5)
                                    delx = delx - 10;
                                if (dely < -5)
                                    dely = dely + 10;
                                else if (dely > 5)
                                    dely = dely - 10;
                                if (delz < -5)
                                    delz = delz + 10;
                                else if (delz > 5)
                                    delz = delz - 10;
//printf("blockIdx %d threadIdx %d part_id %d %f %f %f %f %f %f %f\n", blockIdx.x, threadIdx.x, part_id, r[part_id * 3], r[part_id * 3 + 1], r[part_id *3 + 2], x, y, z);
                                double rr;
                                rr = sqrt(delx * delx + dely * dely + delz * delz);
                                                                if(rr < rc) {
                                    //printf("id %f part_id %f \n", id, part_id);
                                if (id == 3999) {
                                    //printf(" id %d part_id %d rr %f \n", id, part_id, rr);
                                }
                                    double fpair;
                                    double wr;
                                    wr = 1 - rr / rc;
                                    fpair = force_a0 * wr;
                                    double delvx, delvy, delvz;
                                    delvx = v[id*3] - v[part_id*3];
                                    delvy = v[id*3+1] - v[part_id*3+1];
                                    delvz = v[id*3+2] - v[part_id*3+2];
                                    double dot;
                                    dot = (delx * delvx + dely * delvy + delz * delvz) / rr;
                                    fpair -= force_gamma * wr * wr * dot;
                                    //fpair += force_sigma * wr * hiprand_normal_double(&state[id]) * 1 / sqrt(dt);
                                    fpair += force_sigma * wr * hiprand_normal_double(&localState) * 1 / sqrt(dt);

                                    f[id*3] += delx * fpair / rr;
                                    f[id*3+1] += dely * fpair / rr;
                                    f[id*3+2] += delz * fpair / rr; 
                                    //f[part_id*3] -= delx * fpair / rr;
                                    //f[part_id*3+1] -= dely * fpair / rr;
                                    //f[part_id*3+2] -= delz * fpair / rr;
                                    atomicAdd_30(&(f[part_id*3]), -delx * fpair /rr);
                                    atomicAdd_30(&(f[part_id*3+1]), -dely * fpair / rr);
                                    atomicAdd_30(&(f[part_id*3+2]), -delz * fpair / rr);
                               }
                            }
                        }
                //                if (id == 3999) 
                //                    printf("force of each cell is %f %f %f\n", f[id*3], f[id*3+1], f[id*3+2]);

                   }
                }
            }
state[id] = localState;

   }
}


__global__ void post_int(double *v, double *f) {
     int id = threadIdx.x + blockDim.x * blockIdx.x;
    //printf("blockIdx.x %d threadIdx.x %d\n", blockIdx.x, threadIdx.x); 
    if (id < N) {
            //full integration
            int i;
            for (i = 0; i < 3; i++) {
                v[id * 3 + i] += 0.5 * f[id * 3 + i] * dt;
            }
 
 }   
}

void gpu_dpd(double** r, double**f, double** v, const char* select, RanMars* random, OUTPUT_struct* output_info, double len_cell, double len_x, double len_y, double len_z) {

        compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
        //allocate r,f,v on GPU
        int size = N * 3 * sizeof(double);
        double* cpu_r = (double *) malloc(size);
        double* cpu_f = (double *) malloc(size);
        double* cpu_v = (double *) malloc(size);
        int i;
        int count = 0;
        for(i = 0; i < N; i++) {
            cpu_r[count] = r[i][0];
            cpu_f[count] = f[i][0];
            cpu_v[count] = v[i][0];
            cpu_r[count+1] = r[i][1];
            cpu_f[count+1] = f[i][1];
            cpu_v[count+1] = v[i][1];
            cpu_r[count+2] = r[i][2];
            cpu_f[count+2] = f[i][2];
            cpu_v[count+2] = v[i][2];
            count += 3; 
        }
        if (count == 3 * N) {
            printf("count right\n");
            //return;
            double residual_r = 0;
            double residual_f = 0;
            double residual_v = 0;
            int j;
            for (j = 0; j < 3 * N; j++) {
                residual_r += (cpu_r[j] - r[j/3][j%3]);
                residual_f += (cpu_f[j] - f[j/3][j%3]);
                residual_v += (cpu_v[j] - v[j/3][j%3]);
            }
            printf("residual %f %f %f\n", residual_r, residual_f, residual_v);
        }
        double* dev_r;
        double* dev_f;
        double* dev_v;
        hipMalloc((void **) &dev_r, size);
        hipMalloc((void **) &dev_f, size);
        hipMalloc((void **) &dev_v, size);
        hipMemcpy(dev_r, cpu_r, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_f, cpu_f, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_v, cpu_v, size, hipMemcpyHostToDevice);
        int** cell_partc_list_res = output_info->cell_partc_list_res;
        int* part_num = output_info->cell_partc_num_res;
        int cell_num = output_info->cell_num;
        int max_num_part = output_info->col_num;

        int* cell_list = (int *) malloc(sizeof(int) * cell_num * max_num_part);
        int* cell_list_count = (int *) malloc(sizeof(int) * cell_num);
        int j;
        count = 0;
        for (j = 0; j < cell_num; j++) {
            int k;
            for (k = 0; k < max_num_part; k++) {
                cell_list[count++] = cell_partc_list_res[j][k];
            }
            cell_list_count[j] = part_num[j]; 
        }
        double residual_count = 0;
        double residual_list = 0;
        for (j = 0; j < cell_num; j++) {
            residual_count += (cell_list_count[j] - part_num[j]);
            int k;
            for (k = 0; k < max_num_part; k++) {
                residual_list += (cell_list[j * max_num_part + k] - cell_partc_list_res[j][k]);
            }
        }

        printf("residual list %f count %f \n", residual_list, residual_count);
        int* dev_cell_list;
        int* dev_cell_list_count;
        int size_cell_list = sizeof(int) * cell_num * max_num_part;
        hipMalloc((void **) &dev_cell_list, size_cell_list);
        hipMalloc((void **) &dev_cell_list_count, sizeof(int) * cell_num);
        hipMemcpy(dev_cell_list, cell_list, size_cell_list, hipMemcpyHostToDevice);
        hipMemcpy(dev_cell_list_count, cell_list_count, sizeof(int) * cell_num, hipMemcpyHostToDevice);
 
        hiprandState *d_state;
        hipMalloc((void**) &d_state, N);
        int ntimestep = 5000;
        double m = 1.0;
        int blockSize = (int) floor(N/1024 + 1);
        printf("go to call iteration\n");
        cout << "verify f " << verify_f(f) << endl;
        ofstream outputfile;
        outputfile.open("dump_gpu.md", ios::out);
        setup_kernel<<<blockSize,1024>>>(d_state);
        hipDeviceSynchronize();
        for (i = 0; i <= 5000; i++) {
            init<<<blockSize, 1024>>>(dev_r, dev_v, dev_f);
            hipDeviceSynchronize();
            f_clear<<<blockSize, 1024>>>(dev_f);
            hipDeviceSynchronize();
            int *dev_ts;
            hipMalloc((void**) &dev_ts, sizeof(int));
            hipMemcpy(dev_ts, &i, sizeof(int), hipMemcpyHostToDevice);
 
            //setup_kernel<<<blockSize,1024>>>(d_state);
            //hipDeviceSynchronize();
            iteration<<<blockSize, 1024>>>(d_state, dev_r, dev_f, dev_v, dev_cell_list, dev_cell_list_count, max_num_part, dev_ts);
            hipDeviceSynchronize();
            post_int<<<blockSize, 1024>>>(dev_v, dev_f);
            hipMemcpy(cpu_r, dev_r, size, hipMemcpyDeviceToHost);
            hipMemcpy(cpu_v, dev_v, size, hipMemcpyDeviceToHost);
            hipMemcpy(cpu_f, dev_f, size, hipMemcpyDeviceToHost);           
            double res = 0;
            for (int j = 0; j < 3 * N; j++) {
                res += cpu_f[j];
            }
            //cout << " cpu_f is " << res << endl; 
            for (int j = 0; j < N; j++) {
                for (int k = 0; k < 3; k++) {
                    r[j][k] = cpu_r[j * 3 + k];
                    v[j][k] = cpu_v[j * 3 + k];
                    f[j][k] = cpu_f[j * 3 + k];
                }
            }
            //rebuild cell list
            int partc_num_def = 4000;
            output_info =  func_partc_incell_stat(r, partc_num_def, len_cell, len_x, len_y, len_z);
            cell_partc_list_res = output_info->cell_partc_list_res;
            part_num = output_info->cell_partc_num_res;
            cell_num = output_info->cell_num;
            max_num_part = output_info->col_num;
            free(cell_list);
            free(cell_list_count);
            cell_list = (int *) malloc(sizeof(int) * cell_num * max_num_part);
            cell_list_count = (int *) malloc(sizeof(int) * cell_num);

            count = 0;
            for (int j = 0; j < cell_num; j++) {
                int k;
                for (k = 0; k < max_num_part; k++) {
                    cell_list[count++] = cell_partc_list_res[j][k];
                }
                cell_list_count[j] = part_num[j]; 
            }
            hipFree(dev_cell_list);
            hipFree(dev_cell_list_count);
            int size_cell_list = sizeof(int) * cell_num * max_num_part;
            hipMalloc((void **) &dev_cell_list, size_cell_list);
            hipMalloc((void **) &dev_cell_list_count, sizeof(int) * cell_num);

            hipMemcpy(dev_cell_list, cell_list, size_cell_list, hipMemcpyHostToDevice);
            hipMemcpy(dev_cell_list_count, cell_list_count, sizeof(int) * cell_num, hipMemcpyHostToDevice);
 
            if(i % 100 == 0) {
                double ke = computeKE(v);
                cout << i << " temp is " << ke * 2 / (3 * 4000 * 1) << endl;
                writeDump(outputfile, r, f, i);
                //cout << "verify force on gpu at step " << i << " is " << verify_f(f) << endl;

            }

        }
        printf("iteration done\n");
        free(cpu_r);
        free(cpu_f);
        free(cpu_v);
        free(cell_list);
        free(cell_list_count);
        hipFree(dev_r);
        hipFree(dev_f);
        hipFree(dev_v);
        hipFree(dev_cell_list);
        hipFree(dev_cell_list_count);
        hipFree(d_state);
}

void next_func(FILE* fptr) {
    
	char		str_buff[256];
	
	//while (!feof(fptr)){
	//	fscanf(fptr, "%s", str_buff);
	//}
	
	for (int loop=0; loop<14; loop++){
		
		fgets(str_buff, 256, fptr);
		
		printf("%d %s", loop, str_buff);
		
	}
	
}


int load_func(FILE* fptr, double* outptr) {
    
	char		str_buff[256];
	double		b,c,d;
	int			a;
	
	fscanf(fptr, "%d", &a);
	fscanf(fptr, "%d", &a);
	
	if (!feof(fptr)){
		fscanf(fptr, "%lf", &b);
		fscanf(fptr, "%lf", &c);
		fscanf(fptr, "%lf", &d);
		outptr[0] = b;
		outptr[1] = c;
		outptr[2] = d;
		return 1;
	}
	else{
		return 0;
	}
	
}

int main(int argc, char* argv[])
{

    int type_of_device = atoi(argv[1]); // 0 - CPU; 1 - GPU

    const char* select = argv[2];
	FILE*		file_ptr;
	
	char		str_input[5];
	double		result[3];
	int			count;
	int			end;
	int			flag;
    double** partc_pos_res;
    int pos_index;
    int partc_num_def = 4000;
    int i;
	
	// read input position file
	count = 0;
	if((file_ptr = fopen("4000_new.txt","r")) == NULL){
		printf("Cannt open the file!");
		exit(1);
	}
	
	next_func(file_ptr);
	
    // allocate memory for partc_pos_res
    partc_pos_res = (double**)malloc(partc_num_def*sizeof(double*));
    for (i=0; i<partc_num_def; i++){
        partc_pos_res[i] = (double*)malloc(3*sizeof(double));
        
    }

    // fill in the particle positions into partc_pos_res
	while (!feof(file_ptr)){
		// count = count+1;
		flag = load_func(file_ptr, result);
		if (flag){

			// printf("Line:%d, %.2f, %.2f, %.2f\n", count, result[0], result[1], result[2]);
            for (pos_index=0; pos_index<3; pos_index++){
                partc_pos_res[count][pos_index] = result[pos_index];
            }
			
		}
        count = count+1;
		
	}


	fclose(file_ptr);
	//scanf("%d",&end);

    // build cell list
    int     loop;
    unsigned int seed;
    
    //input_cube
    double  len_cell;
    double  len_x, len_y, len_z;
    
    //input_partc
    int     partc_num;
    double**    partc_pos;
    
    int     end_flag;
    
    //output
    OUTPUT_struct*  output_info;
    
    seed = 10;
    srand(seed);
    
    //INPUT information:
    len_cell = 2.0;
    len_x = 10.0;
    len_y = 10.0;
    len_z = 10.0;
    
    // partc_num = 10;
    // partc_pos = (double**)malloc(partc_num*sizeof(double*));
    // for(loop=0; loop<partc_num; loop++){
    //     partc_pos[loop] = (double*)malloc(3*sizeof(double));
    //     partc_pos[loop][0] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_x;
    //     partc_pos[loop][1] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_y;
    //     partc_pos[loop][2] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_z;
        
    // }
    
    output_info =  func_partc_incell_stat(partc_pos_res, partc_num_def, len_cell, len_x, len_y, len_z);
    
    //PRINT basic information
    //if (1){
    //}
    //PRINT output information
    //if (1){
    //    func_print_output_info(output_info);
    //}
    //scanf("%d",&end_flag);
    // finished building cell list

    RanMars * random = new RanMars(34387);
     
    int N = 4000;
    double** r = new double* [N];
    double** v = new double* [N];
    double** f = new double* [N];


    for(int i = 0; i < N; i++) {
        r[i] = new double[3];
        v[i] = new double[3];
        f[i] = new double[3];
    }

    for (int i = 0; i < N; i++) {
        r[i][0] = partc_pos_res[i][0]; 
        r[i][1] = partc_pos_res[i][1];
        r[i][2] = partc_pos_res[i][2];
    }

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < 3; j++) {
            v[i][j] = 0;
            f[i][j] = 0;
        }
    }

    if (!type_of_device) {
        cpu_dpd(r, f, v, select, random, output_info, len_cell, len_x, len_y, len_z);   
    } 
    else {
        gpu_dpd(r, f, v, select, random, output_info,len_cell, len_x, len_y, len_z);  
    }

    //memory release
    for(int i = 0; i < N; i++) {
        delete(r[i]);
        delete(v[i]);
        delete(f[i]);
    }

    delete(r);
    delete(v);
    delete(f);
    delete(random);
    cout << "position is read, f is computed" << endl;
 
}





