#include "hip/hip_runtime.h"
#include	<stdio.h>
#include 	<stdlib.h>
#include 	<string.h>
#include	<math.h>
#include	<iostream>
#include	<fstream>
#include    "sim_run.h"
#include    "calc_neighbor_list.h"
#include    "random_mars.h"
#include    "hip/hip_runtime.h"
#include <map>
#include <vector>
using namespace std;

void cpu_dpd(double** r, double**f, double** v, const char* select, RanMars* random, OUTPUT_struct* output_info, double len_cell, double len_x, double len_y, double len_z) {
        ofstream outputfile;
        outputfile.open("dump.md", ios::out);

        //init force compute
        map<int, vector<int>> cell_list;
        if (!strcmp(select, "sijun"))
            compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
        else if (!strcmp(select, "vector")) {
            int cell_nx = 10 / len_cell + 1;
            int cell_ny = 10 / len_cell + 1;
            int cell_nz = 10 / len_cell + 1;
                    for (int i = 0; i < N; i++) {
                int cellx = (int) r[i][0] / len_cell;
                int celly = (int) r[i][1] / len_cell; 
                int cellz = (int) r[i][2] / len_cell;
                int cellid = cellz + celly * cell_nz + cellx * cell_nz * cell_ny;
                cell_list[cellid].push_back(i); 
            }
            compute_force_vector(r, v, f, random, N, cell_list, len_cell, len_x, len_y, len_z);
        }
        else if (!strcmp(select, "base")) 
            compute_force_std(r, v, f, random, N);

        //writeDump(outputfile, r, v, 0);

        int ntimestep = 5000;
        double m = 1.0;
        for (int i = 0; i <= ntimestep; i++) {

            //half integration
            for(int j = 0; j < N; j++) {
                v[j][0] += 0.5 * f[j][0] * dt;
                v[j][1] += 0.5 * f[j][1] * dt;
                v[j][2] += 0.5 * f[j][2] * dt;

                r[j][0] += v[j][0] * dt;
                r[j][1] += v[j][1] * dt;
                r[j][2] += v[j][2] * dt;
            }
            pbc(r); 
            //force computation
            clear_force(f, N);
            if (!strcmp(select, "sijun"))
                compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
            else if (!strcmp(select, "vector"))
                compute_force_vector(r, v, f, random, N, cell_list, len_cell, len_x, len_y, len_z);
            else if (!strcmp(select, "base"))
                compute_force_std(r, v, f, random, N);

            //full integration
            for(int j = 0; j < N; j++) {
                v[j][0] += 0.5 * f[j][0] * dt;
                v[j][1] += 0.5 * f[j][1] * dt;
                v[j][2] += 0.5 * f[j][2] * dt;
            }
             if(i % 1 == 0) {
                double ke = computeKE(v);
                cout << i << " temp is " << ke * 2 / (3 * 4000 * 1) << endl;
                //writeDump(outputfile, r, v, i);
            }
        }
}
__device__ void compute_force_gpu(int id, double* r, double* f, double* v, RanMars * random, int N, int* cell_list, int* cell_list_count,
double len_cell, double len_x, double len_y, double len_z, int avg_num_cell) {
   double m = 1.0;
    int num_cx, num_cy, num_cz;
    num_cx = (int) floor(len_x/len_cell) + 1;
    num_cy = (int) floor(len_y/len_cell) + 1;
    num_cz = (int) floor(len_z/len_cell) + 1;
    double x = r[id * 3];
    double y = r[id * 3 + 1];
    double z = r[id * 3 + 2];
    int idx = (int) floor(x / len_cell);
    int idy = (int) floor(y / len_cell);
    int idz = (int) floor(z / len_cell);
    // loop through 27 boxes
    for (int l = -1; l < 2; l++) {
        for (int m = -1; m < 2; m++) {
            for (int n = -1; n < 2; n++) {
                int newidz = (idz + n + num_cz) % num_cz;
                int newidy = (idy + m + num_cy) % num_cy;
                int newidx = (idx + l + num_cx) % num_cx;
                int cell_id = newidz + newidy * num_cz + newidx * num_cy * num_cz;
                int num_neigh_particle = cell_list_count[cell_id];
                for (int j = 0; j < num_neigh_particle; j++) {
                    int part_id = cell_list[cell_id * avg_num_cell + j];
                    if (part_id > id) {
                        double delx, dely, delz;
                        delx = x - r[part_id * 3];
                        dely = y - r[part_id * 3 + 1];
                        delz = z - r[part_id * 3 + 2];
                        if (delx < - 5)
                            delx = delx + 10;
                        else if (delx > 5)
                            delx = delx - 10;
                        if (dely < -5)
                            dely = dely + 10;
                        else if (dely > 5)
                            dely = dely - 10;
                        if (delz < -5)
                            delz = delz + 10;
                        else if (delz > 5)
                            delz = delz - 10;
                        double rr;
                        rr = sqrt(delx * delx + dely * dely + delz * delz);
                        if(rr < rc) {
                            double fpair;
                            double wr;
                            wr = 1 - rr / rc;
                            fpair = force_a0 * wr;

                            double delvx, delvy, delvz;
                            delvx = v[id*3] - v[part_id*3];
                            delvy = v[id*3+1] - v[part_id*3+1];
                            delvz = v[id*3+2] - v[part_id*3+2];

                            double dot;
                            dot = (delx * delvx + dely * delvy + delz * delvz) / rr;
                            fpair -= force_gamma * wr * wr * dot;

                            fpair += force_sigma * wr * random->gaussian() * 1 / sqrt(dt);

                            f[id*3] += delx * fpair / rr;
                            f[id*3+1] += dely * fpair / rr;
                            f[id*3+2] += delz * fpair / rr;

                            f[part_id*3] -= delx * fpair / rr;
                            f[part_id*3+1] -= dely * fpair / rr;
                            f[part_id*3+2] -= delz * fpair / rr;

                        }
                    }
                }
            }
        }
    }
}

__global__ void iteration(double* r, double* f, double* v, int* cell_list, int* cell_list_count, RanMars* random, int avg_num_cell) {
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    if (id < N) {
            int i;
            // init integration
            for(i = 0; i < 3; i++) {
                v[id * 3 + i] += 0.5 * f[id * 3 + i] * dt;
                r[id * 3 + i] += v[id * 3 + i] * dt;
            } 
            // do periodic boundary condition
            for (i = 0; i < 3; i++) {
                if (r[id * 3 + i] < 0)
                    r[id * 3 + i] += 10;
                else if (r[id * 3 + i] > 10)
                    r[id * 3 + i] -= 10;
            }
            //force computation
            for (i = 0; i < 3; i++)
                f[id * 3 + i] = 0;
            double len_cell = 2.0;
            double len_x = 10.0;
            double len_y = 10.0;
            double len_z = 10.0;
            compute_force_gpu(id, r, f, v, random, N, cell_list, cell_list_count, len_cell, len_x, len_y, len_z, avg_num_cell);

            //full integration
            for (i = 0; i < 3; i++) {
                v[id * 3 + i] += 0.5 * f[id * 3 + i] * dt;
            }
 
    }
}

void gpu_dpd(double** r, double**f, double** v, const char* select, RanMars* random, OUTPUT_struct* output_info, double len_cell, double len_x, double len_y, double len_z) {

        //allocate r,f,v on GPU
        int size = N * 3 * sizeof(double);
        double* cpu_r = (double *) malloc(size);
        double* cpu_f = (double *) malloc(size);
        double* cpu_v = (double *) malloc(size);
        int i = 0;
        int count = 0;
        for(; i < N; i++) {
            cpu_r[count] = r[i][0];
            cpu_f[count] = f[i][0];
            cpu_v[count] = v[i][0];
            cpu_r[count+1] = r[i][1];
            cpu_f[count+1] = f[i][1];
            cpu_v[count+1] = v[i][1];
            cpu_r[count+2] = r[i][2];
            cpu_f[count+2] = f[i][2];
            cpu_v[count+2] = v[i][2];
            count += 3; 
        }
        double* dev_r;
        double* dev_f;
        double* dev_v;
        size_t pitch;
        hipMalloc((void **) dev_r, size);
        hipMalloc((void **) dev_f, size);
        hipMalloc((void **) dev_v, size);
        hipMemcpy(dev_r, r, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_f, f, size, hipMemcpyHostToDevice);
        hipMemcpy(dev_v, v, size, hipMemcpyHostToDevice);

        // build cell list on cpu 
        int ncell_x = 10 / len_cell;
        int ncell_y = 10 / len_cell;
        int ncell_z = 10 / len_cell;
        int num_cell = ncell_x * ncell_y * ncell_z;
        int avg_num_particle = N / num_cell + 1;
        int* cell_list = (int *) malloc(sizeof(int) * num_cell * avg_num_particle);
        int* cell_list_count = (int *) malloc(sizeof(int) * num_cell);
        for (i = 0; i < num_cell * avg_num_particle; i++) {
            cell_list[i] = 0;
        }
        for (i = 0; i < num_cell; i++) {
            cell_list_count[i] = 0;
        }

        i = 0;
        for(; i < N; i++) {
            int cellx = (int) r[i][0] / len_cell;
            int celly = (int) r[i][1] / len_cell;
            int cellz = (int) r[i][2] / len_cell;
            int cellid = cellz + celly * ncell_z + cellx * ncell_z * ncell_y;
            int row = cellid;
            int col = cell_list_count[cellid];
            int index = row * avg_num_particle + col;
            cell_list[index] = i;
            cell_list_count[i]++; 
        }

        int* dev_cell_list;
        int* dev_cell_list_count;
        int size_cell_list = sizeof(int) * num_cell * avg_num_particle;
        hipMalloc((void **) &dev_cell_list, size_cell_list);
        hipMalloc((void **) &dev_cell_list_count, sizeof(int) * num_cell);
        hipMemcpy(dev_cell_list, cell_list, size_cell_list, hipMemcpyHostToDevice);
        hipMemcpy(dev_cell_list_count, cell_list_count, sizeof(int) * num_cell, hipMemcpyHostToDevice);
 
        ofstream outputfile;
        outputfile.open("dump.md", ios::out);

        //init force compute
        //map<int, vector<int>> cell_list;
        //if (!strcmp(select, "sijun"))
        //    compute_force(r, v, f, random, N, output_info, len_cell, len_x, len_y, len_z);
        //else if (!strcmp(select, "vector")) {
        //    int cell_nx = 10 / len_cell + 1;
        //    int cell_ny = 10 / len_cell + 1;
        //    int cell_nz = 10 / len_cell + 1;
        //            for (int i = 0; i < N; i++) {
        //        int cellx = (int) r[i][0] / len_cell;
        //        int celly = (int) r[i][1] / len_cell; 
        //        int cellz = (int) r[i][2] / len_cell;
        //        int cellid = cellz + celly * cell_nz + cellx * cell_nz * cell_ny;
        //        cell_list[cellid].push_back(i); 
        //    }
        //    compute_force_vector(r, v, f, random, N, cell_list, len_cell, len_x, len_y, len_z);
        //}
        //else if (!strcmp(select, "base")) 
        //    compute_force_std(r, v, f, random, N);

        //writeDump(outputfile, r, v, 0);

        int ntimestep = 5000;
        double m = 1.0;
        for (int i = 0; i <= ntimestep; i++) {
            iteration<<<N/1024+1, N>>>(dev_r, dev_f, dev_v, dev_cell_list, dev_cell_list_count, random, avg_num_particle);
            hipMemcpy(r, dev_r, size, hipMemcpyDeviceToHost);
            //build new cell list
         int ncell_x = 10 / len_cell;
        int ncell_y = 10 / len_cell;
        int ncell_z = 10 / len_cell;
        int num_cell = ncell_x * ncell_y * ncell_z;
        int avg_num_particle = N / num_cell + 1;
        int* cell_list = (int *) malloc(sizeof(int) * num_cell * avg_num_particle);
        int* cell_list_count = (int *) malloc(sizeof(int) * num_cell);
        for (i = 0; i < num_cell * avg_num_particle; i++) {
            cell_list[i] = 0;
        }
        for (i = 0; i < num_cell; i++) {
            cell_list_count[i] = 0;
        }

        i = 0;
        for(; i < N; i++) {
            int cellx = (int) r[i][0] / len_cell;
            int celly = (int) r[i][1] / len_cell;
            int cellz = (int) r[i][2] / len_cell;
            int cellid = cellz + celly * ncell_z + cellx * ncell_z * ncell_y;
            int row = cellid;
            int col = cell_list_count[cellid];
            int index = row * avg_num_particle + col;
            cell_list[index] = i;
            cell_list_count[i]++; 
        }

           
            if(i % 1 == 0) {
                double ke = computeKE(v);
                cout << i << " temp is " << ke * 2 / (3 * 4000 * 1) << endl;
                //writeDump(outputfile, r, v, i);
            }
        }
        free(cpu_r);
        free(cpu_f);
        free(cpu_v);
        free(cell_list);
        free(cell_list_count);
        hipFree(dev_r);
        hipFree(dev_f);
        hipFree(dev_v);
        hipFree(dev_cell_list);
        hipFree(dev_cell_list_count);
}

void next_func(FILE* fptr) {
    
	char		str_buff[256];
	
	//while (!feof(fptr)){
	//	fscanf(fptr, "%s", str_buff);
	//}
	
	for (int loop=0; loop<14; loop++){
		
		fgets(str_buff, 256, fptr);
		
		printf("%d %s", loop, str_buff);
		
	}
	
}


int load_func(FILE* fptr, double* outptr) {
    
	char		str_buff[256];
	double		b,c,d;
	int			a;
	
	fscanf(fptr, "%d", &a);
	fscanf(fptr, "%d", &a);
	
	if (!feof(fptr)){
		fscanf(fptr, "%lf", &b);
		fscanf(fptr, "%lf", &c);
		fscanf(fptr, "%lf", &d);
		outptr[0] = b;
		outptr[1] = c;
		outptr[2] = d;
		return 1;
	}
	else{
		return 0;
	}
	
}

int main(int argc, char* argv[])
{

    int type_of_device = 0; // 0 - CPU; 1 - GPU
    const char* select = argv[1];
	FILE*		file_ptr;
	
	char		str_input[5];
	double		result[3];
	int			count;
	int			end;
	int			flag;
    double** partc_pos_res;
    int pos_index;
    int partc_num_def = 4000;
    int i;
	
	// read input position file
	count = 0;
	if((file_ptr = fopen("4000_new.txt","r")) == NULL){
		printf("Cannt open the file!");
		exit(1);
	}
	
	next_func(file_ptr);
	

    // allocate memory for partc_pos_res
    partc_pos_res = (double**)malloc(partc_num_def*sizeof(double*));
    for (i=0; i<partc_num_def; i++){
        partc_pos_res[i] = (double*)malloc(3*sizeof(double));
        
    }

    // fill in the particle positions into partc_pos_res
	while (!feof(file_ptr)){
		// count = count+1;
		flag = load_func(file_ptr, result);
		if (flag){

			// printf("Line:%d, %.2f, %.2f, %.2f\n", count, result[0], result[1], result[2]);
            for (pos_index=0; pos_index<3; pos_index++){
                partc_pos_res[count][pos_index] = result[pos_index];
            }
			
		}
        count = count+1;
		
	}


	fclose(file_ptr);
	//scanf("%d",&end);

    // build cell list
    int     loop;
    unsigned int seed;
    
    //input_cube
    double  len_cell;
    double  len_x, len_y, len_z;
    
    //input_partc
    int     partc_num;
    double**    partc_pos;
    
    int     end_flag;
    
    //output
    OUTPUT_struct*  output_info;
    
    seed = 10;
    srand(seed);
    
    //INPUT information:
    len_cell = 2.0;
    len_x = 10.0;
    len_y = 10.0;
    len_z = 10.0;
    
    // partc_num = 10;
    // partc_pos = (double**)malloc(partc_num*sizeof(double*));
    // for(loop=0; loop<partc_num; loop++){
    //     partc_pos[loop] = (double*)malloc(3*sizeof(double));
    //     partc_pos[loop][0] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_x;
    //     partc_pos[loop][1] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_y;
    //     partc_pos[loop][2] = (double)(rand()%CONST_MAX_RAND)/(CONST_MAX_RAND-1)*len_z;
        
    // }
    
    
    
    output_info =  func_partc_incell_stat(partc_pos_res, partc_num_def, len_cell, len_x, len_y, len_z);
    
    
    //PRINT basic information
    //if (1){
        
        
    //}
    
    //PRINT output information
    //if (1){
    //    func_print_output_info(output_info);
    //}
    
    //scanf("%d",&end_flag);
    // finished building cell list

    RanMars * random = new RanMars(34387);
     
    int N = 4000;
    double** r = new double* [N];
    double** v = new double* [N];
    double** f = new double* [N];


    for(int i = 0; i < N; i++) {
        r[i] = new double[3];
        v[i] = new double[3];
        f[i] = new double[3];
    }

    for (int i = 0; i < N; i++) {
        r[i][0] = partc_pos_res[i][0]; 
        r[i][1] = partc_pos_res[i][1];
        r[i][2] = partc_pos_res[i][2];
    }

    for(int i = 0; i < N; i++) {
        for(int j = 0; j < 3; j++) {
            v[i][j] = 0;
            f[i][j] = 0;
        }
    }

    if (!type_of_device) {
        cpu_dpd(r, f, v, select, random, output_info, len_cell, len_x, len_y, len_z);   
    } 
    else {
        gpu_dpd(r, f, v, select, random, output_info, len_cell, len_x, len_y, len_z);   
    }

    //memory release
    for(int i = 0; i < N; i++) {
        delete(r[i]);
        delete(v[i]);
        delete(f[i]);
    }

    delete(r);
    delete(v);
    delete(f);
    delete(random);
    cout << "position is read, f is computed" << endl;
 
}





